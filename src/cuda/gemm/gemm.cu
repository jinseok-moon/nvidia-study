#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <utils.h>

static LatencyProfiler profiler;

constexpr float EPS = 1e-5;
#define TIMES 1
#define SIZE 4096

bool check_result(float *ref, float *result, int size, float threshold = 0.01) {
  for (int i = 0; i < size; i++) {
    float diff = abs(result[i] - ref[i]);
    float ref_abs = abs(ref[i]);

    // Use relative error if reference value is not too small
    float relative_error = diff / (ref_abs + EPS);
    if (relative_error > threshold) {
      std::cout << "result[" << i << "] = " << result[i] << " != ref[" << i
                << "] = " << ref[i]
                << " (relative error: " << relative_error * 100 << "%)"
                << std::endl;
      return false;
    }
  }
  return true;
}

// Naive GEMM implementation in CPU
void gemm_cpu(float *A, float *B, float *C, int M, int N, int K) {
#pragma omp parallel for collapse(2)
  for (int m = 0; m < M; m++) {
    for (int n = 0; n < N; n++) {
      float sum = 0;
      for (int k = 0; k < K; k++) {
        sum += A[m * K + k] * B[k * N + n];
      }
      C[m * N + n] = sum;
    }
  }
}

// Naive GEMM implementation in GPU
__global__ void gemm_gpu_0_naive(float *A, float *B, float *C, int M, int N,
                                 int K) {
  int m = blockIdx.x * blockDim.x + threadIdx.x;
  int n = blockIdx.y * blockDim.y + threadIdx.y;
  if (n >= N || m >= M)
    return;

  float sum = 0;
  for (int k = 0; k < K; k++) {
    sum += A[m * K + k] * B[k * N + n];
  }

  C[m * N + n] = sum;
}

__global__ void gemm_gpu_1_dram_coalescing(float *A, float *B, float *C, int M,
                                           int N, int K)
{
  int m = blockIdx.y * blockDim.y + threadIdx.y;
  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n >= N || m >= M)
    return;

  float sum = 0;
  for (int k = 0; k < K; k++) {
    sum += A[m * K + k] * B[k * N + n];
  }

  C[m * N + n] = sum;
}

template <int BLOCKSIZE>
__global__ void gemm_gpu_2_sram_caching(float *A, float *B, float *C, int M,
                                        int N, int K)
{
  int cRow = blockIdx.y;
  int cCol = blockIdx.x;

  A += cRow * BLOCKSIZE * K;                    // (bM,0)
  B += cCol * BLOCKSIZE;                        // (0,bN)
  C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // (bM,bN)

  __shared__ float A_shared[BLOCKSIZE * BLOCKSIZE];
  __shared__ float B_shared[BLOCKSIZE * BLOCKSIZE];

  float sum = 0;
  int threadRow = threadIdx.x / BLOCKSIZE;
  int threadCol = threadIdx.x % BLOCKSIZE;

  for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCKSIZE) {
    A_shared[threadRow * BLOCKSIZE + threadCol] = A[threadRow * K + threadCol];
    B_shared[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];
    __syncthreads();

    A += BLOCKSIZE;
    B += BLOCKSIZE * N;

    for (int dotIdx = 0; dotIdx < BLOCKSIZE; dotIdx++) {
      sum += A_shared[threadRow * BLOCKSIZE + dotIdx] *
             B_shared[dotIdx * BLOCKSIZE + threadCol];
    }

    __syncthreads();
  }
  C[threadRow * N + threadCol] = sum;
}

template <int BM, int BN, int BK, int TM>
__global__ void gemm_gpu_3_sram_1d_tiling(float *A, float *B, float *C, int M,
                                          int N, int K)
{

  int cRow = blockIdx.y;
  int cCol = blockIdx.x;
  A += cRow * BM * K;             // (bM,0)
  B += cCol * BN;                 // (0,bN)
  C += cRow * BM * N + cCol * BN; // (bM,bN)

  __shared__ float A_shared[BM * BK];
  __shared__ float B_shared[BK * BN];

  int threadRow = threadIdx.x / BN;
  int threadCol = threadIdx.x % BN;

  int innerRowA = threadIdx.x / BK;
  int innerColA = threadIdx.x % BK;

  int innerRowB = threadIdx.x / BN;
  int innerColB = threadIdx.x % BN;

  float threadResults[TM] = {0.0};
  for (int bkIdx = 0; bkIdx < K; bkIdx += BK) {
    A_shared[innerRowA * BK + innerColA] = A[innerRowA * K + innerColA];
    B_shared[innerRowB * BN + innerColB] = B[innerRowB * N + innerColB];
    __syncthreads();

    A += BK;
    B += BK * N;

    for (int dotIdx = 0; dotIdx < BK; dotIdx++) {
      float _b = B_shared[dotIdx * BN + threadCol];
      for (int resIdx = 0; resIdx < TM; resIdx++) {
        threadResults[resIdx] +=
            A_shared[(threadRow * TM + resIdx) * BK + dotIdx] * _b;
      }
    }
    __syncthreads();
  }

  for (int resIdx = 0; resIdx < TM; ++resIdx) {
    C[(threadRow * TM + resIdx) * N + threadCol] = threadResults[resIdx];
  }
}

template <int BM, int BN, int BK, int TM, int TN>
__global__ void gemm_gpu_4_sram_2d_tiling(float *A, float *B, float *C, int M,
                                          int N, int K)
{

  int cRow = blockIdx.y;
  int cCol = blockIdx.x;
  A += cRow * BM * K;             // (bM,0)
  B += cCol * BN;                 // (0,bN)
  C += cRow * BM * N + cCol * BN; // (bM,bN)

  int totalResultsBlocktile = BM * BN;
  int numThreadsBlocktile = totalResultsBlocktile / (TM * TN);

  __shared__ float A_shared[BM * BK];
  __shared__ float B_shared[BK * BN];

  int threadRow = threadIdx.x / (BN / TN);
  int threadCol = threadIdx.x % (BN / TN);

  int innerRowA = threadIdx.x / BK;
  int innerColA = threadIdx.x % BK;

  int innerRowB = threadIdx.x / BN;
  int innerColB = threadIdx.x % BN;

  int strideA = numThreadsBlocktile / BK;
  int strideB = numThreadsBlocktile / BN;

  float threadResults[TM * TN] = {0.0};
  float regM[TM] = {0.0};
  float regN[TN] = {0.0};

  for (int bkIdx = 0; bkIdx < K; bkIdx += BK) {
    for (int offset = 0; offset < BM; offset += strideA) {
      A_shared[(innerRowA + offset) * BK + innerColA] =
          A[(innerRowA + offset) * K + innerColA];
    }
    for (int offset = 0; offset < BK; offset += strideB) {
      B_shared[(innerRowB + offset) * BN + innerColB] =
          B[(innerRowB + offset) * N + innerColB];
    }
    __syncthreads();

    A += BK;
    B += BK * N;

    for (int dotIdx = 0; dotIdx < BK; dotIdx++) {
      for (int i = 0; i < TM; i++) {
        regM[i] = A_shared[(threadRow * TM + i) * BK + dotIdx];
      }
      for (int i = 0; i < TN; i++) {
        regN[i] = B_shared[dotIdx * BN + threadCol * TN + i];
      }
      for (int resIdxM = 0; resIdxM < TM; resIdxM++) {
        for (int resIdxN = 0; resIdxN < TN; resIdxN++) {
          threadResults[resIdxM * TN + resIdxN] +=
              regM[resIdxM] * regN[resIdxN];
        }
      }
    }
    __syncthreads();
  }

  for (int resIdxM = 0; resIdxM < TM; resIdxM++) {
    for (int resIdxN = 0; resIdxN < TN; resIdxN++) {
      C[(threadRow * TM + resIdxM) * N + threadCol * TN + resIdxN] =
          threadResults[resIdxM * TN + resIdxN];
    }
  }
}

void launch_gpu_kernel_cublas(float *A, float *B, float *C, int M, int N, int K,
                              hipblasHandle_t handle) {
  float alpha = 1.0f;
  float beta = 0.0f;
  cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B,
                HIP_R_32F, N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N);
}

template <int BLOCKSIZE>
void launch_gpu_kernel_0(float *A, float *B, float *C, int M, int N, int K) {
  dim3 block(BLOCKSIZE, BLOCKSIZE, 1);
  dim3 grid((M + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);
  gemm_gpu_0_naive<<<grid, block>>>(A, B, C, M, N, K);
}

template <int BLOCKSIZE>
void launch_gpu_kernel_1(float *A, float *B, float *C, int M, int N, int K) {
  dim3 block(BLOCKSIZE, BLOCKSIZE, 1);
  dim3 grid((N + BLOCKSIZE - 1) / BLOCKSIZE, (M + BLOCKSIZE - 1) / BLOCKSIZE);
  gemm_gpu_1_dram_coalescing<<<grid, block>>>(A, B, C, M, N, K);
}

template <int BLOCKSIZE>
void launch_gpu_kernel_2(float *A, float *B, float *C, int M, int N, int K) {
  dim3 block(BLOCKSIZE * BLOCKSIZE);
  dim3 grid((N + BLOCKSIZE - 1) / BLOCKSIZE, (M + BLOCKSIZE - 1) / BLOCKSIZE);
  gemm_gpu_2_sram_caching<BLOCKSIZE><<<grid, block>>>(A, B, C, M, N, K);
}

template <int BM, int BN, int BK, int TM>
void launch_gpu_kernel_3(float *A, float *B, float *C, int M, int N, int K) {
  dim3 block((BM * BN) / TM);
  dim3 grid(ceil_div(N, BN), ceil_div(M, BM));
  gemm_gpu_3_sram_1d_tiling<BM, BN, BK, TM><<<grid, block>>>(A, B, C, M, N, K);
}

template <int BM, int BN, int BK, int TM, int TN>
void launch_gpu_kernel_4(float *A, float *B, float *C, int M, int N, int K) {
  dim3 block((BM * BN) / (TM * TN));
  dim3 grid(ceil_div(N, BN), ceil_div(M, BM));
  gemm_gpu_4_sram_2d_tiling<BM, BN, BK, TM, TN>
      <<<grid, block>>>(A, B, C, M, N, K);
}

int main(int argc, char *argv[]) {
  // Default values
  int M = SIZE, N = SIZE, K = SIZE;

  // Parse command line arguments
  if (argc >= 2)
    M = std::atoi(argv[1]);
  if (argc >= 3)
    N = std::atoi(argv[2]);
  if (argc >= 4)
    K = std::atoi(argv[3]);

  std::cout << "Matrix dimensions: M=" << M << ", N=" << N << ", K=" << K
            << std::endl;

  // Initialize matrices
  float *A = (float *)malloc(M * K * sizeof(float));
  float *B = (float *)malloc(K * N * sizeof(float));
  float *C = (float *)malloc(M * N * sizeof(float));
  float *dev_A = nullptr;
  float *dev_B = nullptr;
  float *dev_C = nullptr;
  float *host_C = nullptr;
  hipMalloc((void **)&dev_A, M * K * sizeof(float));
  hipMalloc((void **)&dev_B, K * N * sizeof(float));
  hipMalloc((void **)&dev_C, M * N * sizeof(float));
  host_C = (float *)malloc(M * N * sizeof(float));

  // Initialize matrices with random values
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(0, 1);

  for (int i = 0; i < M * K; i++)
    A[i] = dis(gen);
  for (int i = 0; i < K * N; i++)
    B[i] = dis(gen);
  for (int i = 0; i < M * N; i++)
    C[i] = 0;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipMemcpy(dev_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);
  launch_gpu_kernel_cublas(dev_A, dev_B, dev_C, M, N, K, handle);

  // profiler.benchmark_kernel("CPU_GEMM", [&]() { gemm_cpu(A, B, C, M, N, K); });

  profiler.benchmark_kernel("CUBLAS GEMM", [&]() {
    launch_gpu_kernel_cublas(dev_A, dev_B, dev_C, M, N, K, handle);
  });
  hipMemcpy(C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(host_C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
  check_result(C, host_C, M * N);
  hipMemset(dev_C, 0, M * N * sizeof(float));

  profiler.benchmark_kernel("GPU GEMM 0 NAIVE", [&]() {
    launch_gpu_kernel_0<32>(dev_A, dev_B, dev_C, M, N, K);
  });

  hipMemcpy(host_C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
  check_result(C, host_C, M * N);
  hipMemset(dev_C, 0, M * N * sizeof(float));
  profiler.benchmark_kernel("GPU GEMM 1 MEMORY COALESCING", [&]() {
    launch_gpu_kernel_1<32>(dev_A, dev_B, dev_C, M, N, K);
  });

  hipMemcpy(host_C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
  check_result(C, host_C, M * N);
  hipMemset(dev_C, 0, M * N * sizeof(float));

  profiler.benchmark_kernel("GPU GEMM 2 SMEM", [&]() {
    launch_gpu_kernel_2<32>(dev_A, dev_B, dev_C, M, N, K);
  });
  hipMemcpy(host_C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
  check_result(C, host_C, M * N);
  hipMemset(dev_C, 0, M * N * sizeof(float));

  profiler.benchmark_kernel("GPU GEMM 3 1D TILING", [&]() {
    launch_gpu_kernel_3<64, 64, 8, 8>(dev_A, dev_B, dev_C, M, N, K);
  });
  hipMemcpy(host_C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
  check_result(C, host_C, M * N);
  hipMemset(dev_C, 0, M * N * sizeof(float));

  profiler.benchmark_kernel("GPU GEMM 4 2D TILING", [&]()
                            { launch_gpu_kernel_4<128, 128, 8, 8, 8>(dev_A, dev_B, dev_C, M, N, K); });
  hipMemcpy(host_C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
  check_result(C, host_C, M * N);
  hipMemset(dev_C, 0, M * N * sizeof(float));

  CUDA_CHECK(hipDeviceSynchronize());

  return 0;
}